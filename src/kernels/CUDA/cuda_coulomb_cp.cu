#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_coulomb_cp.h"
#define CHECK(call) { const hipError_t error = call; if (error != hipSuccess) {printf("Error:%s:%d, ", __FILE__, __LINE__); printf("code:%d, reason: %s\n" , error, hipGetErrorString(error)); exit(1); }}

// CUDA Kernel for  Coulomb Cluster-Particle interaction

__global__ void Coulomb_CP_Lagrange (int batches, int interp_points, double *d_source_x, double *d_source_y, double *d_source_z, double *d_cluster_x, double *d_cluster_y, double *d_cluster_z, double *d_cluster_q, double *d_s_charge, double *d_potential){

          int i=threadIdx.x + blockDim.x * blockIdx.x;
          int j=blockDim.x * gridDim.x;
          int p=0;
          double dx, dy,dz,r2,tp;

                for( int k=i; k<interp_points; k +=j){
                     p=i-((floorf(i/(batches)))*(batches));
                     dx= d_cluster_x[k]-d_source_x[p];
                     dy= d_cluster_y[k]-d_source_y[p];
                     dz= d_cluster_z[k]-d_source_z[p];

                      r2=(dx*dx)+(dy*dy)+(dz*dz);
                      d_potential+=d_s_charge(p)/sqrt(r2);



  }
 }



void K_CUDA_Coulomb_CP_Lagrange(int batches, int interp_points, double *source_x, double *source_y, double *source_z, double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q, double *s_charge, double *potential) {

       double *d_source_x,*d_source_y,*d_source_z,*d_cluster_x,*d_cluster_y,*d_cluster_z,*d_cluster_q,*d_s_charge,*d_potential;

       hipMalloc((void **) &d_source_x,batches * interp_points * sizeof(double));
       hipMalloc((void **) &d_source_y,batches * interp_points * sizeof(double));
       hipMalloc((void **) &d_source_z,batches * interp_points * sizeof(double));
       hipMalloc((void **) &d_cluster_x,interp_points * sizeof(double));
       hipMalloc((void **) &d_cluster_y,interp_points * sizeof(double));
       hipMalloc((void **) &d_cluster_z,interp_points * sizeof(double));
       hipMalloc((void **) &d_cluster_q,interp_points * sizeof(double));
       hipMalloc((void **) &d_s_charge,batches * interp_points * sizeof(double));
       hipMalloc((void **) &d_potential,batches * interp_points * sizeof(double));
       hipMemcpy(d_source_x,source_x, batches* interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_source_y,source_y, batches* interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_source_z,source_z, batches* interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_cluster_x,cluster_x, interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_cluster_y,cluster_y, interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_cluster_z,cluster_z, interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_cluster_q,cluster_q, interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_s_charge,s_charge, batches * interp_points * sizeof(double),hipMemcpyHostToDevice);
       hipMemcpy(d_potential,0.0, batches * interp_points * sizeof(double),hipMemcpyHostToDevice);


         Coulomb_CP_Lagrange<<<1,32>>>(batches,interp_points,d_source_x,d_source_y,d_source_z,d_cluster_x,d_cluster_y, d_cluster_z,d_cluster_q,d_s_charge, d_potential);
        hipDeviceSynchronize();
        hipMemcpy(potential,d_potential, batches * interp_points *sizeof(double),hipMemcpyDeviceToHost);
        hipFree(d_source_x);
        hipFree(d_source_y);
        hipFree(d_source_z);
        hipFree(d_cluster_x);
        hipFree(d_cluster_y);
        hipFree(d_cluster_z);
        hipFree(d_cluster_q);
        hipFree(d_s_charge);
        hipFree(d_potential);

        return;

}
