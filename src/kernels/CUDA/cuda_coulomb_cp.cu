#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define CHECK(call) { const hipError_t error = call; if (error != hipSuccess) {printf("Error:%s:%d, ", __FILE__, __LINE__); printf("code:%d, reason: %s\n" , error, hipGetErrorString(error)); exit(1); }}

// CUDA Kernel for  Coulomb Cluster-Particle interaction

__global__ void K_CUDA_Coulomb_CP_Lagrange (int batches, int interp_points, double *source_x, double *source_y, double *source_z, double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q, double *s_charge, double *potential){

          int i=threadIdx.x + blockDim.x * blockIdx.x;
          int j=blockDim.x * gridDim.x;
          int p=0;
          double dx, dy,dz,r2,tp;

                for( int k=i; k<interp_points; k +=j){
                     p=i-((floorf(i/(batches)))*(batches));
                     dx= cluster_x[i]-source_x[p];
                     dy= cluster_y[i]-source_y[p];
                     dz= cluster_z[i]-source_z[p];

                      r2=(dx*dx)+(dy*dy)+(dz*dz);
                      tp=+=s_charge(p)/sqrt(r2);



  }
 }

