#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_cluster.h"


__global__ void clust_int(double *clusterX, double *clusterY, double *clusterZ, double *d_tt,double *d_nodeX,\
                          double *d_nodeY, double *d_nodeZ,double h_xb,double h_yb, double h_zb, double x0, \
                          double y0, double z0, int interpDegreeLim, int interpolationDegree, \
                          int interpolationPointsPerCluster, int startingIndexInClustersArray){


               int i=threadIdx.x + blockDim.x * blockIdx.x;
               for ( int o=i;o<interpDegreeLim; o++){
                    d_tt[o]=cos(o * M_PI / interpolationDegree);
                    d_nodeX[o]=x0 + (d_tt[o] + 1.0)/2.0 * h_xb;
                    d_nodeY[o]=y0 + (d_tt[o] + 1.0)/2.0 * h_yb;
                    d_nodeZ[o]=z0 + (d_tt[o] + 1.0)/2.0 * h_zb;
                  }
               for ( j=i;j<interpolationPointsPerCluster;j++){
                   int k1 = j%(interpolationDegree+1);
                   int kk = (j-k1)/(interpolationDegree+1);
                   int k2 = kk%(interpolationDegree+1);
                   kk=kk-k2;
                   int k3 = kk/ (interpolationDegree+1);
                   clusterX[startingIndexInClustersArray +j]=d_nodeX[k1];
                   clusterY[startingIndexInClustersArray +j]=d_nodeY[k2];
                   clusterZ[startingIndexInClustersArray +j]=d_nodeZ[k3];
                  }
  }




void cluster_interp(double *clusterX, double *clusterY, double *clusterZ, \
                          double h_xb,double h_yb, double h_zb, double x0, \
                          double y0, double z0, int interpDegreeLim, int interpolationDegree, \
                          int interpolationPointsPerCluster, int startingIndexInClustersArray){

    double *d_tt, *d_nodeX, *d_nodeY, *d_nodeZ, *d_clusterX,*d_clusterY,*d_clusterZ;
    double d_xb,d_yb,d_zb,h_xb,h_yb,h_zb,d_x0,d_y0,d_z0;
    hipMalloc((void **) &d_tt, interpDegreeLim * sizeof(double));
    hipMalloc((void **) &d_nodeX, interpDegreeLim * sizeof(double));
    hipMalloc((void **) &d_nodeY, interpDegreeLim * sizeof(double));
    hipMalloc((void **) &d_nodeZ, interpDegreeLim * sizeof(double));
    hipMalloc((void **) &d_clusterX, interpolationPoinstsPerCluster * sizeof(double));
    hipMalloc((void **) &d_clusterY, interpolationPointsPerCluster * sizeof(double));
    hipMalloc((void **) &d_clusterZ, interpolationPointsPerCluster * sizeof(double));
    hipMemcpy(d_tt,0.0, interpDegreeLim * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_nodeX,0.0, interpDegreeLim * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_nodeY,0.0, interpDegreeLim * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_nodeZ,0.0, interpDegreeLim * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_clusterX,clusterX, interpolationPointsPerCluster * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_clusterY,clusterY, interpolationPointsPerCluster * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_clusterZ,clusterZ, interpolationPointsPerCluster * sizeof(double),hipMemcpyHostToDevice);
    clust_int<<<1, 32>>(d_clusterX,d_clusterY,d_clusterZ,d_tt,d_nodeX,d_nodeY,d_nodeZ,h_xb,h_yb,h_zb,x0, \
                        y0,z0,interpDegreeLim,interpolationDegree,interpolationPointsPerCluster, \
                        startingIndexInClustersArray)
    hipDeviceSynchronize();
    hipMemcpy(clusterX,d_clusterX,interpolationPointsPerCluster * sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(clusterY,d_clusterY,interpolationPointsPerCluster * sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(clusterZ,d_clusterZ,interpolationPointsPerCluster * sizeof(double),hipMemcpyDeviceToHost);
    hipFree(d_tt);
    hipFree(d_nodeX);
    hipFree(d_nodeY);
    hipFree(d_nodeZ);
    hipFree(d_clusterX);
    hipFree(d_clusterY);
    hipFree(d_clusterZ);
    return; 
} 
